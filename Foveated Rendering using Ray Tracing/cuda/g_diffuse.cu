#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_cuda.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "device_include/helpers.h"
#include "device_include/prd.h"
#include "device_include/random.h"
#include "device_include/commonStructs.h"
#include "device_include/shared_helper_funcs.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float2, Kd_map_scale, , );

rtDeclareVariable(Matrix4x4, prev_mvp, , );
rtDeclareVariable(Matrix4x4, mvp, , );
rtDeclareVariable(float2, screen, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, eye, , );

rtBuffer<ParallelogramLight>     lights;
rtDeclareVariable(float3, light_position, , );

rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(float3, up, , );               // global up vector
rtDeclareVariable(float3, cam_target, , );

RT_PROGRAM void diffuse()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	const float z1 = rnd(prd_radiance.seed);
	const float z2 = rnd(prd_radiance.seed);

	float3 diffDir; float3 specDir;
	optix::cosine_sample_hemisphere(z1, z2, diffDir);
	specDir = optix::reflect(ray.direction, world_geometric_normal);

	const optix::Onb onb(ffnormal);
	onb.inverse_transform(diffDir);
	const float3 hitpoint = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);

	prd_radiance.origin = hitpoint;
	prd_radiance.direction = diffDir;

	const float3 Kd = make_float3(tex2D(Kd_map, texcoord.x / Kd_map_scale.x, texcoord.y / Kd_map_scale.y));
	prd_radiance.result *= Kd;

	// my self
	prd_radiance.normal = world_geometric_normal;
	prd_radiance.depth_value = make_float3(length(hitpoint - eye));

#ifndef ��Ȧī�޶��
	prd_radiance.reproject_uv = compute_reprojection(hitpoint, prev_mvp, screen);
#else
	prd_radiance.reproject_uv = compute_reprojection_pinhole(hitpoint, eye, cam_target, up, screen);
#endif
	 


	// ���� ���� �׸���
	// ���� ���� �׸���
	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	if (num_lights < 1)
		result = make_float3(1.0f);

	bool isShadow = true;
	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float3 light_pos = light_position + light.v1 + light.v2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		// cast shadow ray
		if (nDl > 0.0f && LnDl > 0.0f)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, 2, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			isShadow = shadow_prd.inShadow;
			//if (!shadow_prd.inShadow)
			//{
			//	const float A = length(cross(light.v1, light.v2));
			//	// convert area based pdf to solid angle
			//	const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
			//	result += light.emission * weight;
			//}
		}
	}

	prd_radiance.radiance = make_float3(1-isShadow);
}