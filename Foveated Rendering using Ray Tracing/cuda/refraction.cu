#include "hip/hip_runtime.h"
#include <optix_cuda.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "device_include/helpers.h"
#include "device_include/prd.h"
#include "device_include/random.h"
#include "device_include/commonStructs.h"
#include "device_include/shared_helper_funcs.h"
#include "device_include/shared_helper_math.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float2, Kd_map_scale, , );

rtDeclareVariable(Matrix4x4, prev_mvp, , );
rtDeclareVariable(float2, screen, , );

rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(int, max_depth, , );

rtBuffer<ParallelogramLight>     lights;
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

/// �߰�
rtDeclareVariable(PerRayData_pathtrace_shadow, prd_shadow, rtPayload, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );

rtDeclareVariable(float, fresnel_exponent, , );
rtDeclareVariable(float, fresnel_minimum, , );
rtDeclareVariable(float, fresnel_maximum, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(int, refraction_maxdepth, , );
rtDeclareVariable(int, reflection_maxdepth, , );
rtDeclareVariable(float3, refraction_color, , );
rtDeclareVariable(float3, reflection_color, , );
rtDeclareVariable(float3, extinction_constant, , );

rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void refraction()
{
	// intersection vectors
	const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
	const float3 i = ray.direction;                                            // incident direction

	const float3 Kd = make_float3(tex2D(Kd_map, texcoord.x / Kd_map_scale.x, texcoord.y / Kd_map_scale.y));
	//prd_radiance.reflectance = Kd;

	// my self
	prd_radiance.normal = n;
	prd_radiance.depth_value = make_float3(t_hit);

	float reflection = 1.0f;
	float3 result = make_float3(0.0f);

	float3 beer_attenuation;
	if (dot(n, ray.direction) > 0) {
		// Beer's law attenuation
		beer_attenuation = exp(extinction_constant * t_hit);
	}
	else {
		beer_attenuation = make_float3(1);
	}

	// refraction
	if (prd_radiance.depth < min(refraction_maxdepth, max_depth))
	{
		float3 t;                                                            // transmission direction
		if (refract(t, i, n, refraction_index))
		{

			// check for external or internal reflection
			float cos_theta = dot(i, n);
			if (cos_theta < 0.0f)
				cos_theta = -cos_theta;
			else
				cos_theta = dot(t, n);

			reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

			float importance = prd_radiance.importance * (1.0f - reflection) * optix::luminance(refraction_color * beer_attenuation);
			if (importance > importance_cutoff) {
				optix::Ray ray(h, t, 1, scene_epsilon);
				PerRayData_radiance refr_prd;
				refr_prd.depth = prd_radiance.depth + 1;
				refr_prd.importance = importance;

				rtTrace(top_object, ray, refr_prd);
				result += (1.0f - reflection) * refraction_color * refr_prd.result;
			}
			else {
				result += (1.0f - reflection) * refraction_color * cutoff_color;
			}
		}
		// else TIR
	}

	// reflection
	if (prd_radiance.depth < min(reflection_maxdepth, max_depth))
	{
		float3 r = reflect(i, n);

		float importance = prd_radiance.importance * reflection * optix::luminance(reflection_color * beer_attenuation);
		if (importance > importance_cutoff) {
			optix::Ray ray(h, r, 1, scene_epsilon);
			PerRayData_radiance refl_prd;
			refl_prd.depth = prd_radiance.depth + 1;
			refl_prd.importance = importance;

			rtTrace(top_object, ray, refl_prd);
			result += reflection * reflection_color * refl_prd.result;
		}
		else {
			result += reflection * reflection_color * cutoff_color;
		}
	}

	result = result * beer_attenuation;

	prd_radiance.result = Kd * result;
	prd_radiance.done = true;
}

RT_PROGRAM void any_hit_shadow()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float nDi = fabs(dot(world_shading_normal, ray.direction));

	prd_shadow.attenuation *= 1.0f - fresnel_schlick(nDi, 5.0f, 1.0f - shadow_attenuation, make_float3(1.0f));
	//prd_shadow.inShadow = true;

	rtIgnoreIntersection();
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	//shading_buffer[launch_index] = bad_color;
}
