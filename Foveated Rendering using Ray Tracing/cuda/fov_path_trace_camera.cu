#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_cuda.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "device_include/helpers.h"
#include "device_include/prd.h"
#include "device_include/random.h"
#include "device_include/shared_helper_funcs.h"

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(Matrix4x4, mvp, , );

rtDeclareVariable(float4, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(float2, gaze, , );

rtBuffer<float4, 2>              shading_buffer;

rtBuffer<float4, 2>              history_buffer;
rtBuffer<float4, 2>              history_cache;
rtBuffer<float4, 2>              weight_buffer;
rtBuffer<float4, 2>              depth_buffer;

rtBuffer<uint3, 2>				 thread_buffer;
rtBuffer<uint3, 2>				 thread_cache;

rtBuffer<float4, 2>              extra_buffer;

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

rtBuffer<float3, 1> gaze_target;
rtDeclareVariable(float3, up, , );               // global up vector
rtDeclareVariable(float3, cam_target, , );
rtDeclareVariable(float, g_apertureSize, , );
rtDeclareVariable(int, diffuse_max_depth, , );

RT_PROGRAM void ray_trace()
{
	size_t2 screen = shading_buffer.size();
	float2 screenf = make_float2(screen);

	uint3 thread_work = thread_buffer[launch_index];
	uint2 thread_uv = make_uint2(thread_work);

	float3 result = make_float3(0.0f);
	float4 c_history = make_float4(0.0f);

	float4 c_weight = weight_buffer[thread_uv];
	uint2 query_uv = make_uint2(round(c_weight.x), round(c_weight.y));

	if (c_weight.z > 0.0f) {
		// ����þ� �� (���� �ȼ���)
		//c_history += gaussian_blur(history_cache, query_uv, screenf);
		c_history = history_cache[query_uv];

		/*float2 query_uv2 = make_float2(c_weight.x, c_weight.y);
		c_history = bilinear(history_cache, query_uv2, screenf);*/
	}

	bool usingRay = thread_work.z;

	//accum_buffer[launch_index] = make_float4(make_float3(thread_work.z), 1.0f);
	//accum_buffer[launch_index] = make_float4(make_float2(thread_uv) / screenf, thread_work.z, 1.0f);
	//accum_buffer[thread_uv] = make_float4(make_float2(c_weight) / screenf * c_weight.z, c_weight.z, 1.0f);
	//accum_buffer[launch_index] = make_float4(make_float2(launch_index) / make_float2(launch_dim), 0.0f, 1.0f);

	if (!usingRay) {
		history_buffer[thread_uv] = c_history;

		shading_buffer[thread_uv] = color_to_accumulated(c_history);

		return;
	}

	// Main render loop. This is not recursive, and for high ray depths
	// will generally perform better than tracing radiance rays recursively
	// in closest hit programs.
	float3 normal = make_float3(0, 0, 0);
	float3 origin = make_float3(0, 0, 0);
	int depth = 0;

	int sqrt_num_samples = 1;
	float2 jitter_scale = 1.0f / screenf / sqrt_num_samples;
	unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;

	do
	{
		uint seed = tea<16>(screen.x*thread_uv.y + thread_uv.x, c_history.w > 0 ? frame : 0);

		float2 pixel = make_float2(thread_uv) / screenf * 2.0f - 1.0f;
		uint x = samples_per_pixel % sqrt_num_samples;
		uint y = samples_per_pixel / sqrt_num_samples;
		float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
		float2 d = pixel + jitter*jitter_scale;

		float4 tmp = make_float4(d, -1.0f, 1.0f);
		tmp = mvp * tmp;
		float3 nearPos = make_float3(tmp) / tmp.w;

		float3 ray_origin = eye;
		float3 ray_direction = normalize(nearPos - eye);

#define ��Ȧī�޶��
		/*uint seed0 = thread_uv.x * (c_history.w > 0 ? frame : 0) + seed;
		uint seed1 = thread_uv.y * (c_history.w > 0 ? frame : 0) + seed;
		pixel = make_float2((float)thread_uv.x / screenf.x, (float)(screenf.y - thread_uv.y) / screenf.y);
		createPinHoleCam(eye, cam_target, up, gaze_target[0], pixel, ray_origin, ray_direction, g_apertureSize, seed0, seed1);*/


		// Ray ����
		PerRayData_radiance prd;
		prd.result = make_float3(0.0f);
		prd.depth = 0;
		prd.seed = seed;
		prd.done = false;
		prd.importance = 1.0f;

		prd.reflectance = make_float3(1.0f);
		// prd.radiance = make_float3( 1.0f );

		// ���� Ʈ���̽�
		optix::Ray ray(ray_origin, ray_direction, /*ray type*/ 1, scene_epsilon);
		rtTrace(top_object, ray, prd);
		result += prd.result;
		seed = prd.seed;

		depth = prd.depth;

	} while (--samples_per_pixel);
	result /= float(sqrt_num_samples*sqrt_num_samples);

	{
		// �����
		result = Uncharted2ToneMapping(result);

		float4 final_result = make_float4(result, 1.0f) + c_history;
		history_buffer[thread_uv] = final_result;

		shading_buffer[thread_uv] = color_to_accumulated(final_result);
	}
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	//shading_buffer[launch_index] = bad_color;
}

