#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_cuda.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "device_include/helpers.h"
#include "device_include/prd.h"
#include "device_include/random.h"
#include "device_include/shared_helper_funcs.h"

using namespace optix;


rtDeclareVariable(float3, prev_eye, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float, eye_variance, , );
rtDeclareVariable(Matrix4x4, mvp, , );
rtDeclareVariable(Matrix4x4, prev_mvp, , );
rtDeclareVariable(float2, gaze, , );
rtDeclareVariable(float, g_apectureSize, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, bbox_min, , );
rtDeclareVariable(float3, bbox_max, , );
rtDeclareVariable(int, max_depth, , );

rtBuffer<float4, 2>              position_buffer;
rtBuffer<float4, 2>              depth_buffer;
rtBuffer<float4, 2>              depth_cache;
rtBuffer<float4, 2>              weight_buffer;

rtBuffer<float4, 2>              normal_buffer;
rtBuffer<float4, 2>              diffuse_buffer;

rtBuffer<uint3, 2>				 thread_buffer;
rtBuffer<uint3, 2>				 thread_cache;
rtBuffer<float4, 2>              extra_buffer;

rtBuffer<float3, 1> gaze_target;

rtTextureSampler<float4, 2> sampling_map;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


RT_PROGRAM void sampling_step()
{
	size_t2 screen = thread_buffer.size();
	float2 screenf = make_float2(screen);

	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame);

	bool usingRay = false;

	// �ʱ�ȭ
	thread_cache[launch_index] = make_uint3(0);

	// ���� ã��
//	float isBoundary = edge_detect(depth_buffer, launch_index, screenf);

	//float4 tmp = make_float4(make_float2(launch_index) / screenf - 0.5f, 0.0f, 1.0f);
	//tmp = mvp * tmp;
	//float3 nearPos = make_float3(tmp) / tmp.w;

	//float3 ray_origin = eye;
	//float3 ray_direction = normalize(nearPos - eye);

	float4 position = position_buffer[launch_index];
	float4 depth = depth_buffer[launch_index];

	float4 weight = weight_buffer[launch_index];
	float2 query_uv = make_float2(weight.x, weight.y);
	uint2 q_uv = launch_index;

	float isValid = 0.0f; float isEdge = 0.0f; float s_depth_grad = 0.0f;
	if (query_uv.x > -1.0f && query_uv.y > -1.0f) {
		if ((0 <= query_uv.x && query_uv.x < screenf.x - 0.5f) &&
			(0 <= query_uv.y && query_uv.y < screenf.y - 0.5f)) {
			
			q_uv = make_uint2(round(query_uv.x), round(query_uv.y));

			float4 prev_depth = depth_cache[q_uv];
			
			//prev_depth = bilinear(depth_cache, query_uv, screenf);
			//position = bilinear(position_buffer, query_uv, screenf);
			//float4 tmp = make_float4(query_uv / screenf - 0.5f, 0.0f, 1.0f);
			//tmp = prev_mvp.inverse() * tmp;
			//float3 nearPos = make_float3(tmp) / tmp.w;

			//float diff = (prev_depth.x - length(make_float3(position) - prev_eye));
			//diff = pow(diff, 2.0f);
			//diff = abs(diff);

			//s_depth_grad = gradient(depth_cache, launch_index, screenf);

			float depth_epsilon = scene_epsilon; /* obj ���Ϻ��� depth �ּ� ��ȭ���� ���õǾ���� */
			isValid = isCacheHit(prev_depth.x, length(make_float3(position) - prev_eye), depth_epsilon);
			//isValid = (abs(diff) < depth_epsilon ? 1.0f : 0.0f);

			//float4 prev_normal = normal_cache[q_uv];
			//float edge = max(dot(make_float3(prev_normal), normalize(-ray_direction) * 0.5f + 0.5f), 0.0f);
			//isEdge = edge < scene_epsilon ? 1.0f : 0.0f;
			//if (edge < scene_epsilon)
			//	isValid = 0.0f;

			//// ĳ�� �̽�
			//if (history_cache[q_uv].w < 1.0f) {
			//	isValid = 0.0f;
			//}
			//if (isBoundary)
			//	isValid = 0.0f;

			// ����� 1���� ���ø� (���Ҷ� �ּ�ó��)
			//isValid = 0.0f;
		}
	}
	
	// My Equation
	float gaze_dist = length(make_float2(launch_index) - gaze) / length(screenf);
	float alpha = ((1.0f / 0.8f) - 1.0f) / pow(g_apectureSize, 2);
	float sample_rate = clamp((1.0f / (alpha * pow(2 * gaze_dist, 2) + 1)), 0.0f, 1.0f);
	
	// Weier et al's Equation
	float p_min = 0.05f, r0 = g_apectureSize, r1 = g_apectureSize * 2.0f;
	sample_rate = gaze_dist < r0 ? 
		1.0f : 
		(gaze_dist > r1 ? 
			p_min : 
			(1.0f-(1.0f - p_min) * ((gaze_dist - r0) / (r1 - r0)))
		);

	//// Fujita et al's Equation
	//sample_rate = pow(gaze_dist*length(screenf) / 300.0f, -2.0f / 3.0f);
	
	// ������������ ����� ���ٸ� ���̸� ���. (usingRay && !isValid || isBoundary)
	//bool usingRay = (rnd(seed) < sample_rate) && (true ? rnd(seed) < 0.5f : (!isValid)) || isBoundary;
	//usingRay = (rnd(seed) < sample_rate);

	// üũ���� �������� ���ø�
	/*usingRay = true;
	if ((launch_index.x / 2) % 2 == 0 && (launch_index.y / 2) % 2 == 0)
		usingRay = false;*/

	// ���μ� �������� ���ø�
	//usingRay = true;
	//if ((launch_index.x / 50) % 2 == 0)
	//	usingRay = false;
	
	

	

	// �α����� ��� ���ø�
	/*uint2 uv = FowardLogPolar(launch_index, gaze, screenf * 0.25f);
	uint2 xy = InverseLogPolar(uv, gaze, screenf * 0.25f);
	usingRay = length(make_float2(launch_index - xy)) < sqrt(length(make_float2(1.5f))) ? true : false;*/

	gaze_target[0] = make_float3(position_buffer[make_uint2(gaze)]);

	int scale = 4;
	uint2 sampling_uv = make_uint2(scale * (launch_index.x / scale), 
		scale * (launch_index.y / scale));
	uint2 scale_launch_size = make_uint2(screen.x / scale, screen.y / scale);

	float3 RG_BY_L = RGBY(diffuse_buffer, sampling_uv);
	//float s_intensity = luminance(diffuse_buffer, launch_index, sampling_uv);
	float gx = gradient_x(diffuse_buffer, sampling_uv, screenf, scale);
	float gy = gradient_y(diffuse_buffer, sampling_uv, screenf, scale);
	float s_orientation = orientation_by_sobel(gx, gy);

	float s_depth = depth_saliency(depth_buffer, sampling_uv, make_uint2(gaze), length(bbox_max - bbox_min) * 0.005f);
	float s_shadow = normal_buffer[sampling_uv].w;
	float s_normal_grad = gradient(normal_buffer, sampling_uv, screenf, scale);
	
	float velocity = length(make_float2(launch_index) - query_uv) * 0.5f;
	if (query_uv.x < 0.0f && query_uv.y < 0.0f)
		velocity = 0.0f;
	float s_velocity = velocity_map(velocity);
	

	float saliency = 0.0f;// max(gradient(normal_buffer, sampling_uv, screenf / scale, scale), gradient(diffuse_buffer, sampling_uv, screenf / scale, scale));
	/*saliency = max(saliency, RG_BY_L.x);
	saliency = max(saliency, RG_BY_L.y);
	saliency = max(saliency, RG_BY_L.z);
	saliency = max(saliency, s_orientation);*/
	//saliency = max(saliency, s_intensity);
	//saliency = max(saliency, s_normal);
	//saliency = max(saliency, gradient_y(diffuse_buffer, sampling_uv, screenf / scale, scale));

	saliency = ((RG_BY_L.x + RG_BY_L.y) / 2.0f + RG_BY_L.z + s_orientation) / 3.0f;
	saliency = max(saliency, s_normal_grad);
	saliency *= s_depth;
	saliency = max(saliency, s_velocity) * s_shadow;

	// ����ũ ��� ���ø�
	usingRay = masked_sampling(launch_index, gaze_dist, g_apectureSize, saliency);


	// Weier ���ø� + saliency
	/*uint k = tea<16>(screen.x*launch_index.y + launch_index.x, 0);
	bool sample_rate_1 = rnd(k) < sample_rate;
	bool sample_rate_2 = rnd(k) < saliency;
	usingRay = sample_rate_1 | sample_rate_2;*/

	thread_buffer[launch_index] = make_uint3(launch_index, usingRay);
	weight_buffer[launch_index] = make_float4(query_uv, isValid, 0.0f);
	//extra_buffer[launch_index] = make_float4(heatmap(saliency), 1.0f);

	//extra_buffer[launch_index] = make_float4(query_uv / screenf, 0.0f, 1.0f);

	//extra_buffer[launch_index] = make_float4(make_float3(usingRay), 1.0f);
	//weight_buffer[launch_index] = make_float4(heatmap(saliency), 0.0f);
}