#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_cuda.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "device_include/helpers.h"
#include "device_include/prd.h"
#include "device_include/random.h"
#include "device_include/shared_helper_funcs.h"

using namespace optix;


rtDeclareVariable(float3, prev_eye, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float, eye_variance, , );
rtDeclareVariable(Matrix4x4, mvp, , );
rtDeclareVariable(Matrix4x4, prev_mvp, , );
rtDeclareVariable(float2, gaze, , );
rtDeclareVariable(float, g_apertureSize, , );

rtDeclareVariable(float4, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(float3, bbox_min, , );
rtDeclareVariable(float3, bbox_max, , );

rtBuffer<float4, 2>              position_buffer;
rtBuffer<float4, 2>              normal_buffer;
rtBuffer<float4, 2>              depth_buffer;
rtBuffer<float4, 2>              diffuse_buffer;
rtBuffer<float4, 2>              weight_buffer;

rtBuffer<float4, 2>              history_buffer;
rtBuffer<float4, 2>              history_cache;

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float3, 1> gaze_target;
rtDeclareVariable(float3, up, , );               // global up vector
rtDeclareVariable(float3, cam_target, , );


__device__ void d_buffer_init() {
	normal_buffer[launch_index] = make_float4(0.0f);
	position_buffer[launch_index] = make_float4(0.0f);
	depth_buffer[launch_index] = make_float4(0.0f);
	diffuse_buffer[launch_index] = make_float4(0.0f);
	weight_buffer[launch_index] = make_float4(0.0f);
	
	history_buffer[launch_index] = make_float4(0.0f);
	history_cache[launch_index] = make_float4(0.0f);
}

RT_PROGRAM void g_buffer_trace()
{
	if (frame < 1) {
		d_buffer_init();
	}
	size_t2 screen = position_buffer.size();
	float2 screenf = make_float2(screen);

	uint seed = tea<16>(screen.x*launch_index.y + launch_index.x, 0);
	//thread_cache[launch_index] = make_uint3(0);

	float4 tmp = make_float4(make_float2(launch_index) / screenf * 2.0f - 1.0f, -1.0f, 1.0f);
	tmp = mvp * tmp;
	float3 nearPos = make_float3(tmp) / tmp.w;

	float3 ray_origin = eye;
	float3 ray_direction = normalize(nearPos - eye);

#define ��Ȧī�޶��
	/*uint seed0 = launch_index.x + seed;
	uint seed1 = launch_index.y + seed;
	float2 pixel = make_float2((float)launch_index.x / screenf.x, (float)(screenf.y - launch_index.y) / screenf.y);
	createPinHoleCam(eye, cam_target, up, gaze_target[0], pixel, ray_origin, ray_direction, g_apertureSize, seed0, seed1);*/

	PerRayData_radiance prd;
	prd.result = make_float3(1.0f);
	prd.depth = 0;
	prd.seed = seed;
	prd.done = false;

	// These represent the current shading state and will be set by the closest-hit or miss program

	// attenuation (<= 1) from surface interaction.
	prd.reflectance = make_float3(1.0f);

	// light from a light source or miss program
	//prd.radiance = make_float3(1.0f);

	// next ray to be traced
	prd.normal = make_float3(0.0f);
	prd.depth_value = make_float3(0.0f);
	prd.reproject_uv = make_float2(-1.0f);

	float3 result = make_float3(0.0f);

	// Main render loop. This is not recursive, and for high ray depths
	// will generally perform better than tracing radiance rays recursively
	// in closest hit programs.

	// ���� Ʈ���̽�
	optix::Ray ray(ray_origin, ray_direction, /*ray type*/ 0, scene_epsilon);
	rtTrace(top_object, ray, prd);

	result += prd.result;// *prd.radiance;
	//result += prd.reflectance * cutoff_color;
	if (prd.done) {
		result += prd.result;// *prd.radiance;
	}

	// tone mapping
	//result = Uncharted2ToneMapping(result);
	position_buffer[launch_index] = make_float4(prd.origin, 1.0f);
	//normal_buffer[launch_index] = make_float4(prd.normal * 0.5f + 0.5f, 1.0f);
	normal_buffer[launch_index] = make_float4(prd.normal * 0.5f + 0.5f, prd.radiance.x);
	depth_buffer[launch_index] = make_float4(prd.depth_value, 1.0f);
	diffuse_buffer[launch_index] = make_float4(result, 1.0f);
	weight_buffer[launch_index] = make_float4(prd.reproject_uv, 0.0f, 1.0f);
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	diffuse_buffer[launch_index] = bad_color;
}

