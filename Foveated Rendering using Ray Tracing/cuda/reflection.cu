#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_cuda.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "device_include/helpers.h"
#include "device_include/prd.h"
#include "device_include/random.h"
#include "device_include/commonStructs.h"
#include "device_include/shared_helper_funcs.h"
#include "device_include/shared_helper_math.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float2, Kd_map_scale, , );

rtDeclareVariable(Matrix4x4, prev_mvp, , );
rtDeclareVariable(float2, screen, , );
rtDeclareVariable(float3, eye, , );

rtBuffer<ParallelogramLight>     lights;
rtDeclareVariable(float3, light_position, , );

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(int, reflection_max_depth, , );

rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(float3, reflectivity_n, , );

#if 1
RT_PROGRAM void reflection()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	const float3 hitpoint = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);

	// my self
	prd_radiance.normal = world_geometric_normal;
	prd_radiance.depth_value = make_float3(length(hitpoint - eye));

	const float3 Kd = make_float3(tex2D(Kd_map, texcoord.x / Kd_map_scale.x, texcoord.y / Kd_map_scale.y));

	// ���� ���� �׸���
	unsigned int num_lights = lights.size();
	float3 shadow_result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		ParallelogramLight light = lights[i];
		const float z1 = rnd(prd_radiance.seed);
		const float z2 = rnd(prd_radiance.seed);
		const float3 light_pos = light_position + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		if (nDl > 0.0f && LnDl > 0.0f)
		{
			// cast shadow ray
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			optix::Ray shadow_ray(hitpoint, L, 2, scene_epsilon, Ldist);
			rtTrace(top_object, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if (fmaxf(light_attenuation) > 0.0f) {
				const float A = length(cross(light.v1, light.v2));

				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				float3 Lc = light.emission * weight * light_attenuation;
				shadow_result += Kd * nDl * Lc;
				//shadow_result += light.emission * weight * light_attenuation;

				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
					shadow_result += Ks * Lc * pow(nDh, phong_exp);
			}
		}
	}
	prd_radiance.reflectance *= Kd * shadow_result;

	float3 result = Kd * shadow_result;
	float3 r = schlick(-dot(ffnormal, ray.direction), reflectivity_n);
	float importance = prd_radiance.importance * optix::luminance(r);

	// reflection ray
	if (importance > importance_cutoff && prd_radiance.depth < reflection_max_depth) {
		PerRayData_radiance refl_prd;
		refl_prd.importance = importance;
		refl_prd.depth = prd_radiance.depth + 1;
		refl_prd.reflectance = make_float3(0.0f); //prd_radiance.reflectance;

		float3 R = reflect(ray.direction, ffnormal);
		optix::Ray refl_ray(hitpoint, R, 1, scene_epsilon);

		rtTrace(top_object, refl_ray, refl_prd);
		result += r * refl_prd.reflectance;// refl_prd.result;
	}

	prd_radiance.result = result;


	//float3 result = make_float3(0.0f);
	////prd_radiance.result = prd_radiance.reflectance * prd_radiance.radiance;

	//// depth �ݺ�
	//if (prd_radiance.done) {
	//	prd_radiance.result += prd_radiance.reflectance;
	//}
	//else if (prd_radiance.depth < reflection_max_depth) {
	//	Ray ray(hitpoint, specDir, 1, scene_epsilon);
	//	PerRayData_radiance refl_prd;
	//	refl_prd.depth = prd_radiance.depth + 1;
	//	refl_prd.result = make_float3(0.0f);
	//	refl_prd.reflectance = prd_radiance.reflectance;

	//	rtTrace(top_object, ray, refl_prd);

	//	//result = refl_prd.result;// refl_prd.reflectance * refl_prd.radiance;
	//	prd_radiance.result += refl_prd.reflectance * refl_prd.result;
	//}
}
#else
RT_PROGRAM void reflection()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 specDir;
	const float3 hitpoint = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);

	prd_radiance.origin = hitpoint;
	prd_radiance.direction = specDir;

	const float3 Kd = make_float3(tex2D(Kd_map, texcoord.x / Kd_map_scale.x, texcoord.y / Kd_map_scale.y));
	prd_radiance.reflectance *= Kd;

	// my self
	prd_radiance.normal = world_geometric_normal;
	prd_radiance.depth_value = make_float3(t_hit);

	// ���� ���� �׸���
	unsigned int num_lights = lights.size();
	float3 shadow_result = make_float3(0.0f);

	if (num_lights < 1)
		shadow_result = make_float3(1.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		ParallelogramLight light = lights[i];
		const float z1 = rnd(prd_radiance.seed);
		const float z2 = rnd(prd_radiance.seed);
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		if (nDl > 0.0f && LnDl > 0.0f)
		{
			// cast shadow ray
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			optix::Ray shadow_ray(hitpoint, L, 2, scene_epsilon, Ldist);
			rtTrace(top_object, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if (fmaxf(light_attenuation) > 0.0f) {
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				shadow_result += light.emission * weight * light_attenuation;
			}
		}
	}
	prd_radiance.radiance = shadow_result;

	prd_radiance.result = shadow_result * Kd;
}
#endif
//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );
RT_PROGRAM void shadow()
{
	current_prd_shadow.attenuation = make_float3(0.0f);
	//current_prd_shadow.inShadow = true;
	rtIgnoreIntersection();
	/*rtTerminateRay();

	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float nDi = fabs(dot(world_shading_normal, ray.direction));

	current_prd_shadow.attenuation *= 1.0f - fresnel_schlick(nDi, 5.0f, 1.0f - shadow_attenuation, make_float3(1.0f));
	current_prd_shadow.inShadow = true;

	rtIgnoreIntersection();*/
}