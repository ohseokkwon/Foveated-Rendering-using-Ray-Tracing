#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_cuda.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "device_include/helpers.h"
#include "device_include/prd.h"
#include "device_include/random.h"
#include "device_include/commonStructs.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float2, Kd_map_scale, , );

rtDeclareVariable(Matrix4x4, prev_mvp, , );
rtDeclareVariable(float2, screen, , );
rtDeclareVariable(float3, eye, , );

rtBuffer<ParallelogramLight>     lights;
rtDeclareVariable(float3, light_position, , );

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(int, diffuse_max_depth, , );

#if 1
RT_PROGRAM void diffuse()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	const float z1 = rnd(prd_radiance.seed);
	const float z2 = rnd(prd_radiance.seed);

	float3 diffDir; float3 specDir;
	optix::cosine_sample_hemisphere(z1, z2, diffDir);
	specDir = optix::reflect(ray.direction, world_geometric_normal);

	const optix::Onb onb(ffnormal);
	onb.inverse_transform(diffDir);
	const float3 hitpoint = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);

	// my self
	prd_radiance.normal = world_geometric_normal;
	prd_radiance.depth_value = make_float3(length(hitpoint - eye));

	const float3 Kd = make_float3(tex2D(Kd_map, texcoord.x / Kd_map_scale.x, texcoord.y / Kd_map_scale.y));

	// ���� ���� �׸���
	unsigned int num_lights = lights.size();
	float3 shadow_result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		ParallelogramLight light = lights[i];
		const float z1 = rnd(prd_radiance.seed);
		const float z2 = rnd(prd_radiance.seed);
		const float3 light_pos = light_position + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		if (nDl > 0.0f && LnDl > 0.0f)
		{
			// cast shadow ray
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			optix::Ray shadow_ray(hitpoint, L, 2, scene_epsilon, Ldist);
			rtTrace(top_object, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if (fmaxf(light_attenuation) > 0.0f) {
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				shadow_result += light.emission * weight * light_attenuation;
			}
		}
	}
	prd_radiance.reflectance = Kd * shadow_result;

	float3 result = Kd * shadow_result;
	//prd_radiance.result = prd_radiance.reflectance * prd_radiance.radiance;

	// depth �ݺ�
	if (prd_radiance.done) {
		result += Kd * shadow_result;
	}
	else if (prd_radiance.depth < diffuse_max_depth-1) {
		Ray refl_ray(hitpoint, diffDir, 1, scene_epsilon);
		PerRayData_radiance refl_prd;
		refl_prd.depth = prd_radiance.depth + 1;
		refl_prd.result = make_float3(0.0f);
		refl_prd.reflectance = make_float3(0.0f);

		rtTrace(top_object, refl_ray, refl_prd);

		//result = refl_prd.result;// refl_prd.reflectance * refl_prd.radiance;
		result += refl_prd.reflectance;
	}

	prd_radiance.result = result;
}
#else
RT_PROGRAM void diffuse()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	const float z1 = rnd(prd_radiance.seed);
	const float z2 = rnd(prd_radiance.seed);

	float3 diffDir; float3 specDir;
	optix::cosine_sample_hemisphere(z1, z2, diffDir);
	specDir = optix::reflect(ray.direction, world_geometric_normal);

	const optix::Onb onb(ffnormal);
	onb.inverse_transform(diffDir);
	const float3 hitpoint = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);

	prd_radiance.origin = hitpoint;
	prd_radiance.direction = diffDir;

	const float3 Kd = make_float3(tex2D(Kd_map, texcoord.x / Kd_map_scale.x, texcoord.y / Kd_map_scale.y));
	prd_radiance.reflectance *= Kd;

	// my self
	prd_radiance.normal = world_geometric_normal;
	prd_radiance.depth_value = make_float3(t_hit);

	// ���� ���� �׸���
	unsigned int num_lights = lights.size();
	float3 shadow_result = make_float3(0.0f);

	if (num_lights < 1)
		shadow_result = make_float3(1.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		ParallelogramLight light = lights[i];
		const float z1 = rnd(prd_radiance.seed);
		const float z2 = rnd(prd_radiance.seed);
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		if (nDl > 0.0f && LnDl > 0.0f)
		{
			// cast shadow ray
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			optix::Ray shadow_ray(hitpoint, L, 2, scene_epsilon, Ldist);
			rtTrace(top_object, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if (fmaxf(light_attenuation) > 0.0f) {
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				shadow_result += light.emission * weight * light_attenuation;
			}
		}
	}
	prd_radiance.radiance = shadow_result;

	prd_radiance.result = shadow_result * Kd;
}
#endif
//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );
RT_PROGRAM void shadow()
{
	current_prd_shadow.attenuation = make_float3(0.0f);
	//current_prd_shadow.inShadow = true;
	rtTerminateRay();
	//rtIgnoreIntersection();
	/*rtTerminateRay();

	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float nDi = fabs(dot(world_shading_normal, ray.direction));

	current_prd_shadow.attenuation *= 1.0f - fresnel_schlick(nDi, 5.0f, 1.0f - shadow_attenuation, make_float3(1.0f));
	current_prd_shadow.inShadow = true;

	rtIgnoreIntersection();*/
}