#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#include <optix_cuda.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "device_include/helpers.h"
#include "device_include/random.h"

using namespace optix;

rtBuffer<uint3, 2>				 thread_buffer;
rtBuffer<uint3, 2>				 thread_cache;

rtDeclareVariable(uint2, sort_idx, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, step, , );
rtDeclareVariable(uint, divide, , );
rtDeclareVariable(float2, gaze, , );

rtBuffer<uint, 1> ray_count;

//__device__ __constant__ uint2 offset[9] = {
//	(+1, -1), (+1, 0), (+1, +1),
//	(+0, -1), (+0, 0), (+0, +1),
//	(-1, -1), (-1, 0), (-1, +1)
//};

__device__ __constant__ uint2 offset[9] = {
	(+1, +0), (+1, +1),
	(+0, +0), (+0, +1),
};
//__device__ __constant__ uint2 offset[25] = {
//	(+2, -2), (+2, -1), (+2, +0), (+2, +1), (+2, +2),
//	(+1, -2), (+1, -1), (+1, +0), (+1, +1), (+1, +2),
//	(+0, -2), (+0, -1), (+0, +0), (+0, +1), (+0, +2),
//	(-1, -2), (-1, -1), (-1, +0), (-1, +1), (-1, +2),
//	(-2, -2), (-2, -1), (-2, +0), (-2, +1), (-2, +2),
//};


RT_PROGRAM void warp_sort()
{
	size_t2 screen = thread_buffer.size();
	uint2 self_coord, partner_coord;

	uint2 uv = make_uint2(0, launch_index.y);
	uint2 end_uv = make_uint2(screen.x - 1, launch_index.y);
	uint3 self, partner;

	if (step == 30) {
		ray_count[0] = 0;
		for (int i = 0; i < screen.y; i++)
			ray_count[0] += thread_cache[make_uint2(0, i)].z;

		return;
	}
	// CSR format comppress
	/*if (step == 0) {
		for (int i = 0; i < screen.x - 1; i++) {
			partner_coord = make_uint2(i, launch_index.y);
			partner = thread_buffer[partner_coord];

			if (partner.z > 0) {
				thread_cache[uv] = partner;
				thread_cache[make_uint2(0, launch_index.y)].z++;
				uv.x++;
			}
			else {
				thread_cache[end_uv] = make_uint3(partner_coord, 0);
				end_uv.x--;
			}
		}
	}
	else if (step == 1) {
		end_uv = make_uint2(screen.x - 1, launch_index.y);
		for (int i = 0; i < screen.x - 1; i++) {
			partner_coord = make_uint2(i, screen.x-1 - launch_index.y);
			partner = thread_cache[partner_coord];
			self = thread_cache[end_uv];

			if (self.z > 0)
				break;
			else if (partner.z > 0) {
				thread_cache[end_uv] = partner;
				thread_cache[partner_coord] = self;
				end_uv.x--;
			}
		}
	}
	else if (step == 2) {
		for (int i = 0; i < screen.y/2 - 1; i++) {
			partner_coord = make_uint2(launch_index.x, i + screen.y/2);
			self_coord = make_uint2(launch_index.x, i);

			partner = thread_cache[partner_coord];
			self = thread_cache[self_coord];

			if (self.z < 1 && partner.z > 0) {
				thread_cache[self_coord] = partner;
				thread_cache[partner_coord] = self;
			}
		}
	}*/

	//// CSR and last edge equalization
	if (step == 0) {
		uv = make_uint2(0, launch_index.y);
		end_uv = make_uint2(screen.x - 1, launch_index.y);

		for (int i = 0; i < screen.x; i++) {
			partner_coord = make_uint2(i, launch_index.y);
			partner = thread_buffer[partner_coord];

			if (partner.z > 0) {
				thread_cache[uv] = partner;
				thread_cache[make_uint2(0, launch_index.y)].z = uv.x+1;
				uv.x++;
			}
			else {
				thread_cache[end_uv] = partner;
				end_uv.x--;
			}
		}
	}
	else if (step == 31) {
		uv = make_uint2(launch_index.x, 0);
		end_uv = make_uint2(launch_index.x, screen.y - 1);

		for (int i = 0; i < screen.y; i++) {
			partner_coord = make_uint2(launch_index.x, i);
			partner = thread_cache[partner_coord];

			if (partner.z > 0) {
				thread_buffer[uv] = partner;
				thread_buffer[make_uint2(launch_index.x, 0)].z = uv.y + 1;
				uv.y++;
			}
			else {
				thread_buffer[end_uv] = partner;
				end_uv.y--;
			}
		}
	}
	else if (step > 0 && step < 10) {
		int start_offset_y = launch_index.y * divide;
		uint3 maxElement = thread_buffer[make_uint2(0, start_offset_y + 0)];
		uint3 minElement = thread_buffer[make_uint2(0, start_offset_y + 0)];
		uint2 maxIdx = make_uint2(0, start_offset_y), minIdx = make_uint2(0, start_offset_y);
		for (int i = 0; i < step * 2; i++) {
			if (maxElement.z < thread_buffer[make_uint2(0, start_offset_y + i)].z) {
				maxIdx = make_uint2(0, start_offset_y + i);
			}
			if (minElement.z > thread_buffer[make_uint2(0, start_offset_y + i)].z) {
				minIdx = make_uint2(0, start_offset_y + i);
			}
		}

		int max_length = thread_buffer[maxIdx].z, min_length = thread_buffer[minIdx].z;
		uint3 temp;
		uint2 max_target, min_target;
		
		while ((max_length - min_length) > 1) {
			// Ÿ������
			max_target = make_uint2(max_length-1, maxIdx.y);
			min_target = make_uint2(min_length+1, minIdx.y);

			// ���� �����
			thread_buffer[min_target].z = 2.0f;
			thread_buffer[max_target].z = 3.0f;

			// �糡�� ���� ��ȯ
			temp = thread_buffer[max_target];
			thread_buffer[max_target] = thread_buffer[min_target];
			thread_buffer[min_target] = temp;

			// ��ȯ�� ���� ���� ����
			thread_buffer[maxIdx].z--;
			thread_buffer[minIdx].z++;

			// �� min/max ����
			maxElement = thread_buffer[maxIdx];
			minElement = thread_buffer[minIdx];
			for (int i = 0; i < step * 2; i++) {
				if (maxElement.z < thread_buffer[make_uint2(0, start_offset_y + i)].z) {
					maxIdx = make_uint2(0, start_offset_y + i);
				}
				if (minElement.z > thread_buffer[make_uint2(0, start_offset_y + i)].z) {
					minIdx = make_uint2(0, start_offset_y + i);
				}
			}
			
			// ���� ã�� min/max�� ���� ã��
			max_length = thread_buffer[maxIdx].z;
			min_length = thread_buffer[minIdx].z;
		}
	}
	//// Log-Polar
	//else if (step == 20) {
	//	float2 screenf = make_float2(screen);

	//	uint2 uv = FowardLogPolar(make_float2(launch_index), gaze, screenf);

	//	partner = thread_buffer[launch_index];
	//	if(partner.z > 0) {
	//		thread_cache[uv] = thread_buffer[launch_index];
	//	}
	//}
	//// Inverse Log-Polar
	//else if (step == 21) {
	//	for (int i = 0; i < screen.x; i++) {
	//		partner_coord = make_uint2(i, launch_index.y);
	//		partner = thread_cache[partner_coord];

	//		if (partner.z > 0) {
	//			thread_buffer[uv] = partner;
	//			thread_buffer[make_uint2(0, launch_index.y)].z = uv.x + 1;
	//			uv.x++;
	//		}
	//		else {
	//			thread_buffer[end_uv] = partner;
	//			end_uv.x--;
	//		}
	//	}
	//}

	// worst case
	//int endYcount = screen.x / 8;
	//int Ycount = 0;
	//uint2 package_warp = make_uint2(0, 0);
	//uint2 package_end = make_uint2(screen.x - 1, screen.y - 1);
	//for (int h = 0; h < screen.y; h++) {
	//	for (int w = 0; w < screen.x; w++) {
	//		thread_cache[make_uint2(w, h)] = make_uint4(0);
	//	}
	//}
	//for (int h = 0; h < screen.y; h++) {
	//	for (int w = 0; w < screen.x; w++) {
	//		uint4 thread_idx = thread_buffer[make_uint2(w, h)];
	//		
	//		if (thread_idx.z > 0) {
	//			thread_cache[package_warp] = thread_idx;
	//			thread_cache[package_warp].z = screen.x;
	//			package_warp.x++;
	//			if (package_warp.x % 8 == 0) {
	//				package_warp.x = Ycount*8;
	//				package_warp.y++;
	//				if (package_warp.y % screen.y == 0) {
	//					Ycount++;
	//					package_warp.x = Ycount * 8;
	//					package_warp.y = 0;
	//				}
	//			}
	//		}
	//		else {
	//			continue;
	//		}
	//	}
	//}

	// nearest block unit compress
	//int w = launch_index.x + blockDim.x;
	//self = thread_buffer[launch_index];
	//uv = make_uint2(0);
	//if (step % 2 == 0) {
	//	for (int y = 0; y < 2; y++) {
	//		for (int x = 0; x < 2; x++) {
	//			if (x == 0 && y == 0)
	//				continue;
	//			for (int i = 0; i < 4; i++) {
	//				// Get
	//				if ((launch_index.x / blockDim.x) % 2 == 0) {
	//					partner_coord = make_uint2(launch_index.x * 2 + x * blockDim.x,
	//						launch_index.y * 2 + y * blockDim.y) + offset[i];
	//					self_coord = launch_index * 2;

	//					if (partner_coord.x < 0.0 || partner_coord.x >= screen.y || partner_coord.y < 0.0 || partner_coord.y >= screen.y)
	//						continue;

	//					partner = thread_buffer[partner_coord];
	//					self = thread_buffer[self_coord];

	//					if (self.z > 1 && partner.z > 0) {
	//						thread_buffer[self_coord] = partner;
	//						thread_buffer[partner_coord] = self;
	//						break;
	//					}
	//				}
	//			}
	//		}
	//	}
	//}
	//else {
	//	for (int y = 0; y < 2; y++) {
	//		for (int x = 0; x < 2; x++) {
	//			if (x == 0 && y == 0)
	//				continue;
	//			for (int i = 0; i < 4; i++) {
	//				// Give
	//				if ((launch_index.x / blockDim.x) % 2 == 1) {
	//					partner_coord = make_uint2(launch_index.x * 2 + x * blockDim.x,
	//						launch_index.y * 2 + y * blockDim.y) + offset[i];
	//					self_coord = launch_index * 2;
	//					if (partner_coord.x < 0.0 || partner_coord.x >= screen.y || partner_coord.y < 0.0 || partner_coord.y >= screen.y)
	//						continue;

	//					partner = thread_buffer[partner_coord];
	//					self = thread_buffer[self_coord];

	//					if (self.z > 0 && partner.z < 1) {
	//						thread_buffer[partner_coord] = self;
	//						thread_buffer[self_coord] = partner;
	//						break;
	//					}
	//				}
	//			}
	//		}
	//	}
	//}

		/*for (int i = 0; i < 9; i++) {
			uv = make_uint2(w, launch_index.y) + offset[i];
			if (uv.x < 0.0 || uv.x >= screen.y || uv.y < 0.0 || uv.y >= screen.y)
				continue;

			partner = thread_buffer[uv];
			if (self.z < 1 && partner.z > 0) {
				thread_buffer[launch_index] = partner;
				thread_buffer[uv] = self;
				break;
			}
		}
	}
	else {
		for (int i = 0; i < 9; i++) {
			uv = make_uint2(w, launch_index.y) + offset[i];
			if (uv.x < 0.0 || uv.x >= screen.y || uv.y < 0.0 || uv.y >= screen.y)
				continue;

			partner = thread_buffer[uv];
			if (self.z > 0 && partner.z < 1) {
				thread_buffer[launch_index] = partner;
				thread_buffer[uv] = self;
				break;
			}
		}
	}*/

	/*uv = make_uint2(w, launch_index.y);
	if ((uv.x >= 0.0 && uv.x < screen.y && uv.y >= 0.0 && uv.y < screen.y)) {
		partner = thread_buffer[uv];
		if (step % 2 == 0) {
			if (self.z < 1 && partner.z > 0) {
				thread_buffer[launch_index] = partner;
				thread_buffer[uv] = self;
			}
		}
		else {
			if (self.z > 0 && partner.z < 1) {
				thread_buffer[launch_index] = partner;
				thread_buffer[uv] = self;
			}
		}
	}*/

	/*self = thread_buffer[launch_index];
	if(step == 0)
		partner_coord = make_uint2(launch_index.x, launch_index.y + screen.x * 0.5f);
	else if (step == 1)
		partner_coord = make_uint2(launch_index.x + screen.x * 0.5f, launch_index.y);
	else if (step == 2)
		partner_coord = make_uint2(launch_index.x + screen.x * 0.5f, launch_index.y + screen.x * 0.5f);

	for (int i = 0; i < 9; i++) {
		partner_coord += offset[i];
		if (partner_coord.x < 0.0 || partner_coord.x >= screen.y || partner_coord.y < 0.0 || partner_coord.y >= screen.y)
			continue;

		partner = thread_buffer[partner_coord];
		if (self.z < 1 && partner.z > 0) {
			thread_buffer[launch_index] = partner;
			thread_buffer[partner_coord] = self;
			break;
		}
	}*/
}